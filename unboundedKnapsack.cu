
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

/* 
 * HackerRank Problem Solving(Intermediate).
 * Dynamic Programming approach with Recursion.
 * 11/11 tests accepted.
 * Time Complexity O(k*n), Space Complexity O(n+k).
 * Pretty common problem uhuh@@
 */

int unboundedKnapsack(int k, vector<int> arr, vector<int> memoi) {
    
    if(memoi[k]!=-1) return memoi[k];
    if(k==0) return 0;
    
    int ret=0;
    int temp=0;
    
    for(size_t i=0; i<arr.size(); ++i){
        if(k>=arr[i]){
            temp=unboundedKnapsack(k-arr[i],arr,memoi)+arr[i];
        }
        if(temp==k){
            memoi[k]=k;
            return k;
        }
        if (temp>ret){
            ret=temp;
        }
    }
    
    memoi[k]=ret;
        
    return ret;
}

int main(){
        
    int t=0,n=0,k=0;
    cin>>t;
    
    for(int x=0; x<t; ++x){
        cin>>n>>k;
        
        vector<int> arr(n);
        vector<int> memoi(k+1,-1); //Use a memoization technique with a vector
        
        for(int j=0; j<n; ++j){
            cin>>arr[j];
        }
        
        cout<<unboundedKnapsack(k,arr,memoi)<<'\n';
    }
    return 0;
}